﻿#include <stdio.h>
#include <memory>
#include <iostream>
#include <vector>
#include <random>
#include <chrono>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"


// ********************************
using data_type = float;

constexpr size_t ROWS{ 10000 };
constexpr size_t COLS{ 10000 };

__device__ constexpr int LAPLACE_KERNEL[3][3]
{
	{0, 1, 0},
	{1, -4, 1},
	{0, 1, 0}
};

constexpr int LAPLACE_KERNEL_SIZE{ 3 };
constexpr int HALF_KERNEL_SIZE{ LAPLACE_KERNEL_SIZE / 2 };

template <typename T>
void cuda_deleter(T* deleted_ptr)
{
	printf("%p released!\n", deleted_ptr);
	hipFree(deleted_ptr);
}

template <typename T>
using device_smart_ptr = std::unique_ptr<T, decltype(&cuda_deleter<T>)>;

template <typename T>
device_smart_ptr<T> smart_device_malloc(size_t size)
{
	T* temp{};
	hipMalloc(&temp, size * sizeof(T));
	return device_smart_ptr<T>{ temp, cuda_deleter<T> };
}

__global__ void laplace_operator_cuda_kernel(data_type* result, const data_type* source, int rows, int cols)
{
	unsigned int x{ blockIdx.x * blockDim.x + threadIdx.x };
	unsigned int y{ blockIdx.y * blockDim.y + threadIdx.y };
	if (x < HALF_KERNEL_SIZE || x >= rows - HALF_KERNEL_SIZE
		|| y < HALF_KERNEL_SIZE || y >= cols - HALF_KERNEL_SIZE)
		return;

	result[y * cols + x]
		= source[(y - 1) * cols + x - 1] * LAPLACE_KERNEL[0][0]
		+ source[(y - 1) * cols + x - 0] * LAPLACE_KERNEL[0][1]
		+ source[(y - 1) * cols + x + 1] * LAPLACE_KERNEL[0][2]
		+ source[(y - 0) * cols + x - 1] * LAPLACE_KERNEL[1][0]
		+ source[(y - 0) * cols + x - 0] * LAPLACE_KERNEL[1][1]
		+ source[(y - 0) * cols + x + 1] * LAPLACE_KERNEL[1][2]
		+ source[(y + 1) * cols + x - 1] * LAPLACE_KERNEL[2][0]
		+ source[(y + 1) * cols + x - 0] * LAPLACE_KERNEL[2][1]
		+ source[(y + 1) * cols + x + 1] * LAPLACE_KERNEL[2][2];
}

void fill_random_data(data_type(&data)[ROWS][COLS])
{
	constexpr int SEED{ 1350 };
	std::ranlux48_base random{ SEED };
	for (size_t i = 1; i < ROWS - 1; i++)
		for (size_t j = 1; j < COLS - 1; j++)
			data[i][j] = random();
}

hipError_t laplace_operator_cuda(data_type* result, const data_type* source, unsigned int rows, unsigned int cols)
{
	const size_t elements_count{ (size_t)rows * cols };
	//cudaError_t device_set_result{ cudaSetDevice(0) };
	device_smart_ptr<data_type> dev_result{ smart_device_malloc<data_type>(elements_count) };
	device_smart_ptr<data_type> dev_source{ smart_device_malloc<data_type>(elements_count) };

	std::cout << "Start copy into GPU memory" << std::endl;
	hipMemcpy(dev_source.get(), source, elements_count * sizeof(data_type), hipMemcpyHostToDevice);
	std::cout << "Copied into GPU" << std::endl;

	std::cout << "Kernel started" << std::endl;
	const dim3 BLOCKS_COUNT{ cols / LAPLACE_KERNEL_SIZE + 1, rows / LAPLACE_KERNEL_SIZE + 1 };
	const dim3 THREADS_COUNT{ LAPLACE_KERNEL_SIZE, LAPLACE_KERNEL_SIZE };
	laplace_operator_cuda_kernel<<<BLOCKS_COUNT, THREADS_COUNT>>>(dev_result.get(), dev_source.get(), rows, cols);
	std::cout << "Kernel finished" << std::endl;

	hipDeviceSynchronize();

	std::cout << "Start copy from GPU memory" << std::endl;
	hipMemcpy(result, dev_result.get(), elements_count * sizeof(data_type), hipMemcpyDeviceToHost);
	std::cout << "Copied from GPU" << std::endl;

	return hipError_t(0);
}
// ********************************

int main()
{
	std::unique_ptr<data_type[]> input_ptr{ new data_type[ROWS * COLS] {} };
	std::unique_ptr<data_type[]> output_ptr{ new data_type[ROWS * COLS] {} };
	data_type(&input)[ROWS][COLS]{ reinterpret_cast<data_type(&)[ROWS][COLS]>(*input_ptr.get()) };
	data_type(&output)[ROWS][COLS]{ reinterpret_cast<data_type(&)[ROWS][COLS]>(*output_ptr.get()) };

	std::cout << "Initialize data" << std::endl;
	fill_random_data(input);
	std::cout << "Initialize finished" << std::endl;

	// расчёт и замеры
	std::cout << "Start calculation" << std::endl;
	std::chrono::steady_clock clock{};
	std::chrono::steady_clock::time_point start{ clock.now() };

	hipError_t cudaStatus = laplace_operator_cuda(output_ptr.get(), input_ptr.get(), ROWS, COLS);

	std::chrono::steady_clock::time_point end{ clock.now() };
	std::cout << "Calculated in: "
		<< std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() << " ms" << std::endl;

	cudaStatus = hipDeviceReset();
	std::cout << "Success!" << std::endl;
	return 0;
}
